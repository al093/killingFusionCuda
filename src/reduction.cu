#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "reduction.cuh"

#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include "helper.cuh"

#include "hipblas.h"

void findAbsMax(hipblasHandle_t handle, float* maxVal, const float * d_array, size_t width, size_t height, size_t depth)
{
    int maxIdx = 0;
    // If cublas was able to find the max value, copy the value from the index in the gpu memory to host memory
    if(hipblasIsamax(handle, width*height*depth, d_array, 1, &maxIdx) == HIPBLAS_STATUS_SUCCESS)
    {
        // cublas has fortran like 1 based indexing
        hipMemcpy(maxVal, d_array+maxIdx-1, sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
        *maxVal = abs(*maxVal);
    }
    else
    {
        std::cout << std::endl <<"[ERROR] CUBLAS encountered an error while calculating the max value";
    }
}
