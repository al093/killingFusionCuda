#include "optimizer.cuh"

#include <iostream>
#include <fstream>
#include <cmath>
#include <Eigen/Geometry>
#include <hip/hip_runtime.h>
#include "convolution.cuh"
#include "divergence.cuh"
#include "helper.cuh"
#include <opencv2/highgui/highgui.hpp>

Optimizer::Optimizer(TSDFVolume* tsdfGlobal, float* initialDeformation, const float alpha, const float wk, const float ws, const size_t gridW, const size_t gridH, const size_t gridD) :
	m_tsdfGlobal(tsdfGlobal),
    m_deformationField(initialDeformation),
    m_alpha(alpha),
	m_wk(wk),
	m_ws(ws),
	m_gridW(gridW), 
	m_gridH(gridH),
	m_gridD(gridD)
{
    allocateMemoryInDevice();
	copyArraysToDevice();
}

void Optimizer::allocateMemoryInDevice()
{
	// Allocate deformation field
	hipMalloc(&m_d_deformationField, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	// Allocate kernels
	hipMalloc(&m_d_kernelDx, (27) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_kernelDy, (27) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_kernelDz, (27) * sizeof(float)); CUDA_CHECK;
	// Allocate gradients
	hipMalloc(&m_d_dx, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_dy, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_dz, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	// Allocate divergence
	hipMalloc(&m_d_div, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
}

void Optimizer::copyArraysToDevice()
{
	hipMemcpy(m_d_kernelDx, m_kernelDxCentralDiff, (27) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(m_d_kernelDy, m_kernelDyCentralDiff, (27) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(m_d_kernelDz, m_kernelDzCentralDiff, (27) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
}

Optimizer::~Optimizer()
{
	hipFree(m_d_deformationField); CUDA_CHECK;
	hipFree(m_d_kernelDx); CUDA_CHECK;
	hipFree(m_d_kernelDy); CUDA_CHECK;
	hipFree(m_d_kernelDz); CUDA_CHECK;
	hipFree(m_d_dx); CUDA_CHECK;
	hipFree(m_d_dy); CUDA_CHECK;
	hipFree(m_d_dz); CUDA_CHECK;
	hipFree(m_d_div); CUDA_CHECK;
}

void Optimizer::optimize(float* optimDeformation, TSDFVolume* tsdfLive)
{
	// Initialize variables
	float currentMaxVectorUpdate = 1000000.0;

	// TODO: compute gradient of tsdfLive

	// TODO: compute hessian of tsdfLive

	do
	{
		// Copy necessary arrays from host to device
		hipMemcpy(m_d_deformationField, m_deformationField, (m_gridW * m_gridH * m_gridD) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;

		// TODO: compute laplacians of the deformation field
		
		// TODO: compute divergence of deformation field
		computeDivergence(m_d_div, m_d_deformationField, m_kernelDxCentralDiff, m_kernelDyCentralDiff, m_kernelDzCentralDiff, 1, m_gridW, m_gridH, m_gridD);
		// TODO: interpolate on gradient of tsdfLive
		
		// TODO: interpolate on hessian of tsdfLive

		// TODO: compute dEdata term

		// TODO: compute dEkilling term

		// TODO: compute dElevel_set term

		// TODO compute dEnon_rigid

		// TODO: update new state of the deformation field

	} while (currentMaxVectorUpdate > MAX_VECTOR_UPDATE_THRESHOLD);
}

void Optimizer::computeDivergence(float* divOut, const float* deformationIn, const float *kernelDx, const float *kernelDy, const float *kernelDz, int kradius, int w, int h, int d)
{

	// Compute gradients for the deformation field
	computeConvolution3D(m_d_dx, deformationIn, kernelDx, kradius, w, h, d);
    computeConvolution3D(m_d_dy, deformationIn, kernelDy, kradius, w, h, d);
	computeConvolution3D(m_d_dz, deformationIn, kernelDz, kradius, w, h, d);
	// Sum the three gradient components
	computeDivergence3DCuda(divOut, m_d_dx, m_d_dy, m_d_dz, w, h, d);
}
