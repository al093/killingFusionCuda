#include "optimizer.cuh"

#include <iostream>
#include <fstream>
#include <cmath>
#include <Eigen/Geometry>
#include <hip/hip_runtime.h>
#include "convolution.cuh"
#include "divergence.cuh"
#include "helper.cuh"
#include <opencv2/highgui/highgui.hpp>

Optimizer::Optimizer(TSDFVolume* tsdfGlobal, float* initialDeformationU, float* initialDeformationV, float* initialDeformationW, const float alpha, const float wk, const float ws, const size_t gridW, const size_t gridH, const size_t gridD) :
	m_tsdfGlobal(tsdfGlobal),
    m_deformationFieldU(initialDeformationU),
    m_deformationFieldV(initialDeformationV),
    m_deformationFieldW(initialDeformationW),
    m_alpha(alpha),
	m_wk(wk),
	m_ws(ws),
	m_gridW(gridW), 
	m_gridH(gridH),
	m_gridD(gridD)
{
    allocateMemoryInDevice();
	copyArraysToDevice();
}

void Optimizer::allocateMemoryInDevice()
{
	// Allocate deformation field
	hipMalloc(&m_d_deformationFieldU, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_deformationFieldV, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_deformationFieldW, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	// Allocate kernels
	hipMalloc(&m_d_kernelDx, (27) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_kernelDy, (27) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_kernelDz, (27) * sizeof(float)); CUDA_CHECK;
	// Allocate gradients
	hipMalloc(&m_d_dx, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_dy, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	hipMalloc(&m_d_dz, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
	// Allocate divergence
	hipMalloc(&m_d_div, (m_gridW * m_gridH * m_gridD) * sizeof(float)); CUDA_CHECK;
}

void Optimizer::copyArraysToDevice()
{
	hipMemcpy(m_d_kernelDx, m_kernelDxCentralDiff, (27) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(m_d_kernelDy, m_kernelDyCentralDiff, (27) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(m_d_kernelDz, m_kernelDzCentralDiff, (27) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
}

Optimizer::~Optimizer()
{
	hipFree(m_d_deformationFieldU); CUDA_CHECK;
	hipFree(m_d_deformationFieldV); CUDA_CHECK;
	hipFree(m_d_deformationFieldW); CUDA_CHECK;
	hipFree(m_d_kernelDx); CUDA_CHECK;
	hipFree(m_d_kernelDy); CUDA_CHECK;
	hipFree(m_d_kernelDz); CUDA_CHECK;
	hipFree(m_d_dx); CUDA_CHECK;
	hipFree(m_d_dy); CUDA_CHECK;
	hipFree(m_d_dz); CUDA_CHECK;
	hipFree(m_d_div); CUDA_CHECK;
}

void Optimizer::optimize(float* optimDeformationU, float* optimDeformationV, float* optimDeformationW, TSDFVolume* tsdfLive)
{
	// Initialize variables
	float currentMaxVectorUpdate = 0.01;

	// TODO: compute gradient of tsdfLive

	// TODO: compute hessian of tsdfLive

	do
	{
		// Copy necessary arrays from host to device
		hipMemcpy(m_d_deformationFieldU, m_deformationFieldU, (m_gridW * m_gridH * m_gridD) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
		hipMemcpy(m_d_deformationFieldV, m_deformationFieldV, (m_gridW * m_gridH * m_gridD) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
		hipMemcpy(m_d_deformationFieldW, m_deformationFieldW, (m_gridW * m_gridH * m_gridD) * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;

		// TODO: compute laplacians of the deformation field
		
		// TODO: compute divergence of deformation field
		computeDivergence(m_d_div, m_d_deformationFieldU, m_d_deformationFieldV, m_d_deformationFieldW, m_d_kernelDx, m_d_kernelDy, m_d_kernelDz, 1, m_gridW, m_gridH, m_gridD);
		// TODO: interpolate on gradient of tsdfLive
		
		// TODO: interpolate on hessian of tsdfLive

		// TODO: compute dEdata term

		// TODO: compute dEkilling term

		// TODO: compute dElevel_set term

		// TODO compute dEnon_rigid

		// TODO: update new state of the deformation field

	} while (currentMaxVectorUpdate > MAX_VECTOR_UPDATE_THRESHOLD);
}

void Optimizer::computeDivergence(float* divOut, const float* deformationInU, const float* deformationInV, const float* deformationInW, const float *kernelDx, const float *kernelDy, const float *kernelDz, int kradius, int w, int h, int d)
{

	// Compute gradients for the deformation field
	computeConvolution3D(m_d_dx, deformationInU, kernelDx, kradius, w, h, d);
    computeConvolution3D(m_d_dy, deformationInV, kernelDy, kradius, w, h, d);
	computeConvolution3D(m_d_dz, deformationInW, kernelDz, kradius, w, h, d);
	// Sum the three gradient components
	computeDivergence3DCuda(divOut, m_d_dx, m_d_dy, m_d_dz, w, h, d);
}
