#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "energyDerivatives.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"

__global__
void computeDataTermDerivativeKernel(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW, 
                                    const float *d_phiNDeformed, const float *d_phiGlobal,
                                    const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                                    const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;

        float scalar = (d_phiNDeformed[idx] - d_phiGlobal[idx]);

        d_dEdataU[idx] += scalar*d_gradPhiNDeformedX[idx];
        d_dEdataV[idx] += scalar*d_gradPhiNDeformedY[idx];
        d_dEdataW[idx] += scalar*d_gradPhiNDeformedZ[idx];
    }
}

__global__
void computeLevelSetDerivativeKernel(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW, 
                               const float *d_hessPhiXX, const float *d_hessPhiXY, const float *d_hessPhiXZ,
                               const float *d_hessPhiYY, const float *d_hessPhiYZ, const float *d_hessPhiZZ,
                               const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                               const bool *d_mask, const float ws,
                               const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;

        //if(d_mask[idx])
        {
            float gradNorm = d_gradPhiNDeformedX[idx]*d_gradPhiNDeformedX[idx] + d_gradPhiNDeformedY[idx]*d_gradPhiNDeformedY[idx] + d_gradPhiNDeformedZ[idx]*d_gradPhiNDeformedZ[idx];
            gradNorm = sqrt(gradNorm);

            float scalar = ws*(gradNorm - 1.0/0.05)/(gradNorm+0.00001);

            d_dEdataU[idx] += scalar*(d_hessPhiXX[idx]*d_gradPhiNDeformedX[idx] + d_hessPhiXY[idx]*d_gradPhiNDeformedY[idx] + d_hessPhiXZ[idx]*d_gradPhiNDeformedZ[idx]);
            d_dEdataV[idx] += scalar*(d_hessPhiXY[idx]*d_gradPhiNDeformedX[idx] + d_hessPhiYY[idx]*d_gradPhiNDeformedY[idx] + d_hessPhiYZ[idx]*d_gradPhiNDeformedZ[idx]);
            d_dEdataW[idx] += scalar*(d_hessPhiXZ[idx]*d_gradPhiNDeformedX[idx] + d_hessPhiYZ[idx]*d_gradPhiNDeformedY[idx] + d_hessPhiZZ[idx]*d_gradPhiNDeformedZ[idx]);
        }
    }
}

__global__
void computeMotionRegularizerDerivativeKernel(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW,
                                              const float *d_lapU, const float *d_lapV, const float *d_lapW,
                                              const float *d_divX, const float *d_divY, const float *d_divZ,
                                              const float wk, const float gamma,
                                              const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;
        d_dEdataU[idx] += -2.0*wk*d_lapU[idx] -2.0*wk*gamma*d_divX[idx];
        d_dEdataV[idx] += -2.0*wk*d_lapV[idx] -2.0*wk*gamma*d_divY[idx];
        d_dEdataW[idx] += -2.0*wk*d_lapW[idx] -2.0*wk*gamma*d_divZ[idx];
    }
}

__global__
void addArrayKernel(float* d_arrayA, const float* d_arrayB, const float scalar, const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;
        d_arrayA[idx] += scalar*d_arrayB[idx];
    }
}

__global__
void addWeightedArrayKernel(float* arrayOut, float* weightOut, const float* arrayIn1, const float* arrayIn2, const float* weight1, const float* weight2, const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;
        float sumWeights = weight1[idx] + weight2[idx];
        if (arrayIn1[idx] == -1 && arrayIn2[idx] == -1)
        {
            arrayOut[idx] = -1;
        }
        else
        {
            arrayOut[idx] = (weight1[idx]*arrayIn1[idx] + weight2[idx]*arrayIn2[idx]) / sumWeights;
        }
        weightOut[idx] = sumWeights;
    }

}

__global__
void multiplyArraysKernel(float* arrayOut, const float* arrayIn1, const float* arrayIn2, const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;
        arrayOut[idx] = arrayIn1[idx] * arrayIn2[idx];
    }

}

__global__
void thresholdArrayKernel(float* arrayOut, const float* arrayIn, float threshold, const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;
        if (arrayIn[idx] < threshold)
        {
            arrayOut[idx] = 0.0f;
        }
    }

}

void computeDataTermDerivative(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW,
                               const float *d_phiNDeformed, const float *d_phiGlobal,
                               const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                               const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeDataTermDerivativeKernel <<<grid, blockSize>>> (d_dEdataU, d_dEdataV, d_dEdataW, 
                                                     d_phiNDeformed, d_phiGlobal,
                                                     d_gradPhiNDeformedX, d_gradPhiNDeformedY, d_gradPhiNDeformedZ,
                                                     width, height, depth);
}

void computeLevelSetDerivative(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW, 
                               const float *d_hessPhiXX, const float *d_hessPhiXY, const float *d_hessPhiXZ,
                               const float *d_hessPhiYY, const float *d_hessPhiYZ, const float *d_hessPhiZZ,
                               const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                               const bool* d_mask, const float wk,
                               const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);
    
    computeLevelSetDerivativeKernel <<<grid, blockSize>>> (d_dEdataU, d_dEdataV, d_dEdataW, 
                                                           d_hessPhiXX, d_hessPhiXY, d_hessPhiXZ,
                                                           d_hessPhiYY, d_hessPhiYZ, d_hessPhiZZ,
                                                           d_gradPhiNDeformedX, d_gradPhiNDeformedY, d_gradPhiNDeformedZ,
                                                           d_mask, wk,
                                                           width, height, depth);
}

void computeMotionRegularizerDerivative(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW,
                                        const float *d_lapU, const float *d_lapV, const float *d_lapW,
                                        const float *d_divX, const float *d_divY, const float *d_divZ,
                                        const float ws, const float gamma,
                                        const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);
    
    computeMotionRegularizerDerivativeKernel <<<grid, blockSize>>> (d_dEdataU, d_dEdataV, d_dEdataW,
                                                                    d_lapU, d_lapV, d_lapW,
                                                                    d_divX, d_divY, d_divZ,
                                                                    ws, gamma,
                                                                    width, height, depth);
}

void addArray(float* d_arrayA, const float* d_arrayB, const float scalar,
              const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    addArrayKernel <<<grid, blockSize>>> (d_arrayA, d_arrayB, scalar,
                                          width, height, depth);
}

void addWeightedArray(float* arrayOut, float* weightOut, const float* arrayIn1, const float* arrayIn2, const float* weight1, const float* weight2, const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    addWeightedArrayKernel <<<grid, blockSize>>> (arrayOut, weightOut, arrayIn1, arrayIn2, weight1, weight2, width, height, depth);
}

void multiplyArrays(float* arrayOut, const float* arrayIn1, const float* arrayIn2, const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    multiplyArraysKernel <<<grid, blockSize>>> (arrayOut, arrayIn1, arrayIn2, width, height, depth);
}

void thresholdArray(float* arrayOut, const float* arrayIn, const float threshold, const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    thresholdArrayKernel <<<grid, blockSize>>> (arrayOut, arrayIn, threshold, width, height, depth);
}
