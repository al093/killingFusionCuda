#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// Authors: Alok Vermaal, Alok.Verma@cs.tum.edu
//          Julio Oscanoa, julio.oscanoa@tum.de
//          Miguel Trasobares, miguel.trasobares@tum.de
// Supervisors: Robert Maier, robert.maier@in.tum.de
//              Christiane Sommer, sommerc@in.tum.de
// Computes the energy derivatives for Killing fusion
// ########################################################################
#include "energyDerivatives.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"

__global__
void computeDataTermDerivativeKernel(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW, 
                                    const float *d_phiNDeformed, const float *d_phiGlobal, const bool *d_mask,
                                    const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                                    const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;
        //if(d_mask[idx])
        {
            float scalar = (d_phiNDeformed[idx] - d_phiGlobal[idx]);

            d_dEdataU[idx] += scalar*d_gradPhiNDeformedX[idx];
            d_dEdataV[idx] += scalar*d_gradPhiNDeformedY[idx];
            d_dEdataW[idx] += scalar*d_gradPhiNDeformedZ[idx];
        }
    }
}

__global__
void computeLevelSetDerivativeKernel(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW, 
                               const float *d_hessPhiXX, const float *d_hessPhiXY, const float *d_hessPhiXZ,
                               const float *d_hessPhiYY, const float *d_hessPhiYZ, const float *d_hessPhiZZ,
                               const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                               const bool *d_mask, const float ws, const float tsdfGradScale, const float voxelSize,
                               const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;

        if(d_mask[idx])
        {
            float gradNorm = d_gradPhiNDeformedX[idx]*d_gradPhiNDeformedX[idx] + d_gradPhiNDeformedY[idx]*d_gradPhiNDeformedY[idx] + d_gradPhiNDeformedZ[idx]*d_gradPhiNDeformedZ[idx];
            gradNorm = sqrt(gradNorm);

            //grad norm is divided by the voxel size, to get gradient in meters.
            //we dont have unit gradient, its scaled gradient because tsdf computation was done like that
            float scalar = ws*(gradNorm - (voxelSize/tsdfGradScale))/(gradNorm+0.00001);
            d_dEdataU[idx] += scalar*(d_hessPhiXX[idx]*d_gradPhiNDeformedX[idx] + d_hessPhiXY[idx]*d_gradPhiNDeformedY[idx] + d_hessPhiXZ[idx]*d_gradPhiNDeformedZ[idx]);
            d_dEdataV[idx] += scalar*(d_hessPhiXY[idx]*d_gradPhiNDeformedX[idx] + d_hessPhiYY[idx]*d_gradPhiNDeformedY[idx] + d_hessPhiYZ[idx]*d_gradPhiNDeformedZ[idx]);
            d_dEdataW[idx] += scalar*(d_hessPhiXZ[idx]*d_gradPhiNDeformedX[idx] + d_hessPhiYZ[idx]*d_gradPhiNDeformedY[idx] + d_hessPhiZZ[idx]*d_gradPhiNDeformedZ[idx]);
        }
    }
}

__global__
void computeMotionRegularizerDerivativeKernel(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW,
                                              const float *d_lapU, const float *d_lapV, const float *d_lapW,
                                              const float *d_divX, const float *d_divY, const float *d_divZ,
                                              const bool *d_mask, const float wk, const float gamma,
                                              const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;
        float scalar = 1.0;
        //if(d_mask[idx])
        {
            d_dEdataU[idx] += scalar*((-2.0*wk*d_lapU[idx] -2.0*wk*gamma*d_divX[idx]));
            d_dEdataV[idx] += scalar*((-2.0*wk*d_lapV[idx] -2.0*wk*gamma*d_divY[idx]));
            d_dEdataW[idx] += scalar*((-2.0*wk*d_lapW[idx] -2.0*wk*gamma*d_divZ[idx]));
        }
    }
}

void computeDataTermDerivative(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW,
                               const float *d_phiNDeformed, const float *d_phiGlobal,
                               const bool *d_mask,
                               const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                               const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeDataTermDerivativeKernel <<<grid, blockSize>>> (d_dEdataU, d_dEdataV, d_dEdataW, 
                                                     d_phiNDeformed, d_phiGlobal,
                                                     d_mask, d_gradPhiNDeformedX, d_gradPhiNDeformedY, d_gradPhiNDeformedZ,
                                                     width, height, depth);
}

void computeLevelSetDerivative(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW, 
                               const float *d_hessPhiXX, const float *d_hessPhiXY, const float *d_hessPhiXZ,
                               const float *d_hessPhiYY, const float *d_hessPhiYZ, const float *d_hessPhiZZ,
                               const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                               const bool* d_mask, const float wk, const float tsdfGradScale, const float voxelSize,
                               const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeLevelSetDerivativeKernel <<<grid, blockSize>>> (d_dEdataU, d_dEdataV, d_dEdataW, 
                                                           d_hessPhiXX, d_hessPhiXY, d_hessPhiXZ,
                                                           d_hessPhiYY, d_hessPhiYZ, d_hessPhiZZ,
                                                           d_gradPhiNDeformedX, d_gradPhiNDeformedY, d_gradPhiNDeformedZ,
                                                           d_mask, wk, tsdfGradScale, voxelSize,
                                                           width, height, depth);
}

void computeMotionRegularizerDerivative(float *d_dEdataU, float *d_dEdataV, float *d_dEdataW,
                                        const float *d_lapU, const float *d_lapV, const float *d_lapW,
                                        const float *d_divX, const float *d_divY, const float *d_divZ,
                                        const bool *d_mask, const float ws, const float gamma,
                                        const size_t width, const size_t height, const size_t depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);
    
    computeMotionRegularizerDerivativeKernel <<<grid, blockSize>>> (d_dEdataU, d_dEdataV, d_dEdataW,
                                                                    d_lapU, d_lapV, d_lapW,
                                                                    d_divX, d_divY, d_divZ,
                                                                    d_mask, ws, gamma,
                                                                    width, height, depth);
}