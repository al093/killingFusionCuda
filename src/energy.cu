#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// Authors: Alok Vermaal, Alok.Verma@cs.tum.edu
//          Julio Oscanoa, julio.oscanoa@tum.de
//          Miguel Trasobares, miguel.trasobares@tum.de
// Supervisors: Robert Maier, robert.maier@in.tum.de
//              Christiane Sommer, sommerc@in.tum.de
// Computes the energy terms for Killing fusion
// ########################################################################
#include "energy.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"

#include "hipblas.h"

__global__
void computeDataEnergyKernel(float *d_dataEnergyArray, 
                             const float *d_phiNDeformed, const float *d_phiGlobal,
                             const bool* d_mask,
                             const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;
        //if(d_mask[idx])
        {
            float diff = d_phiNDeformed[idx] - d_phiGlobal[idx];
            d_dataEnergyArray[idx] = diff * diff;
        }
        //else
        //{
        //    d_dataEnergyArray[idx] = 0.0;
        //}
    }
}

__global__
void computeLevelSetEnergyKernel(float *d_levelSetEnergyArray,
                                 const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                                 const bool* d_mask, const float ws, const float tsdfGradScale, const float voxelSize,
                                 const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;

        if (d_mask[idx])
        {
            float norm = sqrt(d_gradPhiNDeformedX[idx]*d_gradPhiNDeformedX[idx] + d_gradPhiNDeformedY[idx]*d_gradPhiNDeformedY[idx] + d_gradPhiNDeformedZ[idx]*d_gradPhiNDeformedZ[idx]);
            float temp = (tsdfGradScale*norm/voxelSize) - 1.0; 
            d_levelSetEnergyArray[idx] = ws * 0.5 * temp * temp;
        }
        else
        {
            d_levelSetEnergyArray[idx] = 0.0f;
        }
    }
}

__global__
void computeKillingEnergyKernel(float *d_killingEnergyArray, const float gamma,
                                const float* d_dux, const float* d_duy, const float* d_duz,
                                const float* d_dvx, const float* d_dvy, const float* d_dvz,
                                const float* d_dwx, const float* d_dwy, const float* d_dwz,
                                const bool* d_mask, const float wk,
                                const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x < width && y < height && z < depth)
    {
        size_t idx = x + y*width + z*width*height;
        //if(d_mask[idx])
        {
            d_killingEnergyArray[idx] = (1.0+gamma)*(d_dux[idx]*d_dux[idx] + d_dvy[idx]*d_dvy[idx] + d_dwz[idx]*d_dwz[idx])+
                                          d_duy[idx]*d_duy[idx] + d_duz[idx]*d_duz[idx] +
                                          d_dvx[idx]*d_dvx[idx] + d_dvz[idx]*d_dvz[idx] +
                                          d_dwx[idx]*d_dwx[idx] + d_dwy[idx]*d_dwy[idx] +
                                          2.0*gamma*(d_duy[idx]*d_dvx[idx] + d_duz[idx]*d_dwx[idx] + d_dwy[idx]*d_dvz[idx]);
            d_killingEnergyArray[idx] = wk*d_killingEnergyArray[idx];
        }
        //else
        //{
        //    d_killingEnergyArray[idx] = 0.0f;
        //}
    }
}

void computeDataEnergy(float *dataEnergy, const float *d_phiNDeformed, const float *d_phiGlobal,
                       const bool* d_mask,
                       const size_t width, const size_t height, const size_t depth)
{
    float* d_dataEnergyArray;
    hipMalloc(&d_dataEnergyArray, (width * height * depth) * sizeof(float)); CUDA_CHECK;

    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeDataEnergyKernel <<<grid, blockSize>>> (d_dataEnergyArray, 
                                                   d_phiNDeformed, d_phiGlobal,
                                                   d_mask,
                                                   width, height, depth);
    CUDA_CHECK;

    // Create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Calculate the sum of the energy
    hipblasSasum(handle, width*height*depth, d_dataEnergyArray, sizeof(float), dataEnergy);
	  *dataEnergy = 0.5 * *dataEnergy;
    hipblasDestroy(handle);
    hipFree(d_dataEnergyArray);
}

void computeLevelSetEnergy(float *levelSetEnergy,
                           const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                           const bool* d_mask, const float ws, const float tsdfGradScale, const float voxelSize,
                           const size_t width, const size_t height, const size_t depth)
{
    float* d_levelSetEnergyArray;
    hipMalloc(&d_levelSetEnergyArray, (width * height * depth) * sizeof(float)); CUDA_CHECK;

    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeLevelSetEnergyKernel <<<grid, blockSize>>> (d_levelSetEnergyArray, 
                                                       d_gradPhiNDeformedX, d_gradPhiNDeformedY, d_gradPhiNDeformedZ,
                                                       d_mask, ws, tsdfGradScale, voxelSize,
                                                       width, height, depth);
    CUDA_CHECK;

    // Create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Calculate the sum of the energy
    hipblasSasum(handle, width*height*depth, d_levelSetEnergyArray, sizeof(float), levelSetEnergy);

    // Free cuda memory and cublas handle
    hipblasDestroy(handle);
    hipFree(d_levelSetEnergyArray);
}

void computeKillingEnergy(float *killingEnergy, const float gamma,
                          const float* d_dux, const float* d_duy, const float* d_duz,
                          const float* d_dvx, const float* d_dvy, const float* d_dvz,
                          const float* d_dwx, const float* d_dwy, const float* d_dwz,
                          const bool* d_mask, const float wk,
                          const size_t width, const size_t height, const size_t depth)
{
    float* d_killingEnergyArray;
    hipMalloc(&d_killingEnergyArray, (width * height * depth) * sizeof(float)); CUDA_CHECK;

    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeKillingEnergyKernel <<<grid, blockSize>>> (d_killingEnergyArray, gamma,
                                                      d_dux, d_duy, d_duz,
                                                      d_dvx, d_dvy, d_dvz,
                                                      d_dwx, d_dwy, d_dwz,
                                                      d_mask, wk,
                                                      width, height, depth);
    CUDA_CHECK;

    // Create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Calculate the sum of the energy
    hipblasSasum(handle, width*height*depth, d_killingEnergyArray, sizeof(float), killingEnergy);

    // Free cuda memory and cublas handle
    hipblasDestroy(handle);
    hipFree(d_killingEnergyArray);
}
