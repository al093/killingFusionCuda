#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "energy.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"

#include "hipblas.h"

__global__
void computeDataEnergyKernel(float *d_dataEnergyArray, 
                             const float *d_phiNDeformed, const float *d_phiGlobal,
                             const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;
        d_dataEnergyArray[idx] = pow((d_phiNDeformed[idx] - d_phiGlobal[idx]),2);
    }
}

__global__
void computeLevelSetEnergyKernel(float *d_levelSetEnergyArray,
                                 const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                                 const size_t width, const size_t height, const size_t depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;
        float norm = sqrt(pow(d_gradPhiNDeformedX[idx], 2) + pow(d_gradPhiNDeformedY[idx], 2) + pow(d_gradPhiNDeformedZ[idx], 2));
        d_levelSetEnergyArray[idx] = 0.5 * pow((norm - 1), 2);
    }
}





void computeDataEnergy(float *dataEnergy, const float *d_phiNDeformed, const float *d_phiGlobal,
                       const size_t width, const size_t height, const size_t depth)
{
    float* d_dataEnergyArray;
    hipMalloc(&d_dataEnergyArray, (width * height * depth) * sizeof(float)); CUDA_CHECK;

    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeDataEnergyKernel <<<grid, blockSize>>> (d_dataEnergyArray, 
                                                   d_phiNDeformed, d_phiGlobal,
                                                   width, height, depth);
    CUDA_CHECK;

    // create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //calculate the sum of the energy
    hipblasSasum(handle, width*height*depth, d_dataEnergyArray, sizeof(float), dataEnergy);
    hipblasDestroy(handle);
    hipFree(d_dataEnergyArray);
}

void computeLevelSetEnergy(float *levelSetEnergy,
                           const float *d_gradPhiNDeformedX, const float *d_gradPhiNDeformedY, const float *d_gradPhiNDeformedZ,
                           const size_t width, const size_t height, const size_t depth)
{
    float* d_levelSetEnergyArray;
    hipMalloc(&d_levelSetEnergyArray, (width * height * depth) * sizeof(float)); CUDA_CHECK;

    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    computeLevelSetEnergyKernel <<<grid, blockSize>>> (d_levelSetEnergyArray, 
                                                       d_gradPhiNDeformedX, d_gradPhiNDeformedY, d_gradPhiNDeformedZ,
                                                       width, height, depth);
    CUDA_CHECK;

    // create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //calculate the sum of the energy
    hipblasSasum(handle, width*height*depth, d_levelSetEnergyArray, sizeof(float), levelSetEnergy);

    //free cuda memory and cublas handle
    hipblasDestroy(handle);
    hipFree(d_levelSetEnergyArray);
}