#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "visualization.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"
#include <stdlib.h>
#include <fstream>
#include <string>

void getSlice(float* sliceOut, const float* gridIn, const size_t sliceInd, const size_t w, const size_t h)
{
  for(int i = 0; i < w*h; i++)
  {
    sliceOut[i] = gridIn[i + (w*h) * sliceInd];
  }
}

void plotSlice(const float* d_array, const size_t z, const std::string imageTitle, const size_t posX, const size_t posY, const size_t w, const size_t h, const size_t d)
{
    float* h_array = new float[h * w * d];
    float* slice = new float[h * w];
    hipMemcpy(h_array, d_array, (h * w * d) * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
   /* int sizes[] = {(int) w, (int) h, (int) d};
    cv::Mat mat3D(3, sizes, CV_32FC1, cv::Scalar(0));*/
    cv::Mat matSlice(h, w, CV_32F);
    getSlice(slice, h_array, z, w, h);
    convertLayeredToMat(matSlice, slice);
    //getSliceFromMat(mat3D, z, matSlice);
    // Normalize the slice
    double min, max;
    cv::minMaxLoc(matSlice, &min, &max);
    cv::resize(matSlice, matSlice, cv::Size(), 4, 4);
    showImage(imageTitle, (matSlice - min) / (max - min), posX, posY);

    delete[] slice;
    delete[] h_array;
}

void plotVectorField(const float* d_u, const float* d_v, const float* d_w, const float* d_sdf, const size_t sliceZval,
                     const std::string sFileU, const std::string sFileV, const std::string sFileW, const std::string sFileSdf,
                     const std::string sPlotName, const int frameNumber,
                     const size_t width, const size_t height, const size_t depth)
{
    
    float* u = new float[width*height*depth];
    float* v = new float[width*height*depth];
    float* w = new float[width*height*depth];
    float* sdf = new float[width*height*depth];

    hipMemcpy(u, d_u, (width*height*depth) * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(v, d_v, (width*height*depth) * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(w, d_w, (width*height*depth) * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(sdf, d_sdf, (width*height*depth) * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    // save the u , v and w to disk (./bin/result)
    std::ofstream outU(sFileU);
    std::ofstream outV(sFileV);
    std::ofstream outW(sFileW);
    std::ofstream outSdf(sFileSdf);

    //the storing should be done in such a way that python reshape can reshape it correctly
    for (size_t idx = sliceZval*width*height; idx<(sliceZval+1)*width*height; ++idx) {
      outU << u[idx] << " ";
      outV << v[idx] << " ";
      outW << w[idx] << " ";
      outSdf << sdf[idx] << " ";
    }

    outU.close();
    outV.close();
    outW.close();
    outSdf.close();

    delete[] u;
    delete[] v;
    delete[] w;
    delete[] sdf;

    //call python script to plot the quiver plot
    if(system(NULL))
    {
        std::string command = "python ../src/quiverPlot2D.py " +
                               sFileU + " " + sFileV + " " + sFileW + " " +
                               sFileSdf + " " + sPlotName + " " + std::to_string(frameNumber);
        system(command.c_str());
    }
    else
        std::cout<<"\nUnable to access the command prompt/ terminal. Will not be able to show deformation quiver plot";
}
