#include "hip/hip_runtime.h"
#include "interpolator.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include "helper.cuh"

Interpolator::Interpolator(float* h_phi, int width, int height, int depth)
{
    uploadToTextureMemory(h_phi, width, height, depth);
}


Interpolator::~Interpolator(){
    freeTextureMemory();
}

__global__
void interpolate3DKernel(float *d_outputValues, hipTextureObject_t tex, const float *d_u, const float *d_v, const float *d_w, int width, int height, int depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    float fx = x;
    float fy = y;
    float fz = z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;
        // Add 0.5 to get the actual values at the voxels' centers (voxel size 1x1x1) 
        d_outputValues[idx] = tex3D<float>(tex, fx + d_u[idx] + 0.5, fy + d_v[idx] + 0.5, fz + d_w[idx] + 0.5);
    }
}

void Interpolator::uploadToTextureMemory(float* d_grid, int w, int h, int d)
{
    // Define channel format descriptor
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    // Set the grid size
    hipExtent extent;
    extent.width = w;
    extent.height = h;
    extent.depth = d;

    // Allocate 3D cuda array
    hipMalloc3DArray(&cuArray_grid, &desc, extent);
    CUDA_CHECK;

    // Copy from device memory to CudaArray in device memory
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)d_grid, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = cuArray_grid;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.extent = extent;
    hipMemcpy3D(&copyParams);
    CUDA_CHECK;

    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array  = cuArray_grid;
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;
    hipCreateTextureObject(&texGrid, &texRes, &texDescr, NULL);
    CUDA_CHECK;
}

void Interpolator::freeTextureMemory()
{
    hipFreeArray(cuArray_grid); CUDA_CHECK;
	hipDestroyTextureObject(texGrid); CUDA_CHECK;
}


void Interpolator::interpolate3D(float *d_gridInterpolated, const float *d_u, const float *d_v, const float *d_w, int width, int height, int depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    interpolate3DKernel <<<grid, blockSize>>> (d_gridInterpolated, texGrid, d_u, d_v, d_w, width, height, depth);
}
