#include "hip/hip_runtime.h"
///Cuda Functions to calculate the energy derivatives and also helper function for using 3D texture memory.
#include "interpolator.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include "helper.cuh"




Interpolator::Interpolator(float* h_phi, int width, int height, int depth)
{
    uploadToTextureMemory(h_phi, width, height, depth);
}


Interpolator::~Interpolator(){
    freeTextureMemory();
}

__global__
void interpolate3DKernel(float *d_outputValues, hipTextureObject_t tex, const float *d_u, const float *d_v, const float *d_w, int width, int height, int depth)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    float fx = x;
    float fy = y;
    float fz = z;

    if(x<width && y<height && z<depth)
    {
        size_t idx = x + y*width + z*width*height;
        //Remember!! to always add 0.5, the voxels have actual values at their centers, and the size of a voxel is 1x1x1, so need to add .5, .5, .5 for center 
        d_outputValues[idx] = tex3D<float>(tex, fx + d_u[idx] + 0.5, fy + d_v[idx] + 0.5, fz + d_w[idx] + 0.5);
    }
}


///use this method to bind Texture memory to Cuda array.
///TODO currently testing for only one 3D voxel grid
void Interpolator::uploadToTextureMemory(float* h_phi, int w, int h, int d)
{
    hipArray *cuArray_phi;
    //define channel format descriptor
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    //set the grid size
    hipExtent extent;
    extent.width = w;
    extent.height = h;
    extent.depth = d;

    //define and allocate 3D cuda array
    hipMalloc3DArray(&cuArray_phi, &desc, extent);
    CUDA_CHECK;

    //copy from host memory to CudaArray in device memory
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)h_phi, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = cuArray_phi;
    copyParams.kind = hipMemcpyHostToDevice;
    copyParams.extent = extent;
    hipMemcpy3D(&copyParams);
    CUDA_CHECK;

    
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(float));
    resDesc.resType = hipResourceTypeLinear;
         // linear interpolation

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.normalizedCoords = false;                      // access with normalized phiture coordinates
    texDesc.addressMode[0] = hipAddressModeClamp;   // wrap phiture coordinates
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;

/*
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
*/
//-----------------------------------------------------------------------------------------
    //define channel format descriptor
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    //set the grid size
    hipExtent extent;
    extent.width = w;
    extent.height = h;
    extent.depth = d;

    //define and allocate 3D cuda array
    hipMalloc3DArray(&cuArray_phi, &desc, extent);
    CUDA_CHECK;

    //copy from host memory to CudaArray in device memory
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)h_phi, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = cuArray_phi;
    copyParams.kind = hipMemcpyHostToDevice;
    copyParams.extent = extent;
    hipMemcpy3D(&copyParams);
    CUDA_CHECK;

    //set texture parameters
    phi.normalized = false;                      // access with normalized phiture coordinates
    phi.filterMode = hipFilterModeLinear;      // linear interpolation
    phi.addressMode[0] = hipAddressModeClamp;   // wrap phiture coordinates
    phi.addressMode[1] = hipAddressModeClamp;
    phi.addressMode[2] = hipAddressModeClamp;

    // bind array to 3D texture
    hipBindTextureToArray(phi, cuArray_phi, desc);
    CUDA_CHECK;

//-----------------------------------------------------------------------------------------
    // BEGIN WEB CODE
       //hiprand Random Generator (needs compiler link -lcurand)
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen,1235ULL+i);
        hiprandGenerateUniform(gen, d_NoiseTest, cubeSizeNoiseTest);//writing data to d_NoiseTest
        hiprandDestroyGenerator(gen);

        //hipArray Descriptor
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        //cuda Array
        hipArray *d_cuArr;
        checkCudaErrors(hipMalloc3DArray(&d_cuArr, &channelDesc, make_hipExtent(SizeNoiseTest*sizeof(float),SizeNoiseTest,SizeNoiseTest), 0));
        hipMemcpy3DParms copyParams = {0};


        //Array creation
        copyParams.srcPtr   = make_hipPitchedPtr(d_NoiseTest, SizeNoiseTest*sizeof(float), SizeNoiseTest, SizeNoiseTest);
        copyParams.dstArray = d_cuArr;
        copyParams.extent   = make_hipExtent(SizeNoiseTest,SizeNoiseTest,SizeNoiseTest);
        copyParams.kind     = hipMemcpyDeviceToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));
        //Array creation End

        hipResourceDesc    texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array  = d_cuArr;
        hipTextureDesc     texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.addressMode[0] = hipAddressModeClamp;   // clamp
        texDescr.addressMode[1] = hipAddressModeClamp;
        texDescr.addressMode[2] = hipAddressModeClamp;
        texDescr.readMode = hipReadModeElementType;
        checkCudaErrors(hipCreateTextureObject(&texNoise[i], &texRes, &texDescr, NULL));}

void Interpolator::freeTextureMemory()
{
    /*hipUnbindTexture(phi);
    hipFreeArray(cuArray_phi);*/
}


void Interpolator::interpolate3D(float *d_phiInterpolated, const float *d_u, const float *d_v, const float *d_w, int width, int height, int depth)
{
    dim3 blockSize(32, 8, 1);
    dim3 grid = computeGrid3D(blockSize, width, height, depth);

    interpolate3DKernel <<<grid, blockSize>>> (d_phiInterpolated, tex, d_u, d_v, d_w, width, height, depth);
}
